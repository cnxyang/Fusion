#include "hip/hip_runtime.h"
#include "RenderScene.h"
#include "ParallelScan.h"

struct Fusion {

	DeviceMap map;
	float invfx, invfy;
	float fx, fy, cx, cy;
	float minDepth, maxDepth;
	int cols, rows;
	Matrix3f Rview;
	Matrix3f RviewInv;
	float3 tview;

	uint* noVisibleBlocks;

	PtrStep<float4> nmap;
	PtrStep<float> depth;
	PtrStep<uchar3> rgb;

	__device__ inline float2 project(float3& pt3d) {
		float2 pt2d;
		pt2d.x = fx * pt3d.x / pt3d.z + cx;
		pt2d.y = fy * pt3d.y / pt3d.z + cy;
		return pt2d;
	}

	__device__ inline float3 unproject(int& x, int& y, float& z) {
		float3 pt3d;
		pt3d.z = z;
		pt3d.x = z * (x - cx) * invfx;
		pt3d.y = z * (y - cy) * invfy;
		return Rview * pt3d + tview;
	}

	__device__ inline bool CheckVertexVisibility(float3 pt3d) {
		pt3d = RviewInv * (pt3d - tview);
		if (pt3d.z < 1e-3f)
			return false;
		float2 pt2d = project(pt3d);

		return pt2d.x >= 0 && pt2d.y >= 0 &&
			   pt2d.x < cols && pt2d.y < rows &&
			   pt3d.z >= minDepth && pt3d.z <= maxDepth;
	}

	__device__ inline bool CheckBlockVisibility(const int3& pos) {

		float scale = DeviceMap::blockWidth;
		float3 corner = pos * scale;
		if (CheckVertexVisibility(corner))
			return true;
		corner.z += scale;
		if (CheckVertexVisibility(corner))
			return true;
		corner.y += scale;
		if (CheckVertexVisibility(corner))
			return true;
		corner.x += scale;
		if (CheckVertexVisibility(corner))
			return true;
		corner.z -= scale;
		if (CheckVertexVisibility(corner))
			return true;
		corner.y -= scale;
		if (CheckVertexVisibility(corner))
			return true;
		corner.x -= scale;
		corner.y += scale;
		if (CheckVertexVisibility(corner))
			return true;
		corner.x += scale;
		corner.y -= scale;
		corner.z += scale;
		if (CheckVertexVisibility(corner))
			return true;
		return false;
	}

	__device__ inline void CreateBlocks() {

		int x = blockIdx.x * blockDim.x + threadIdx.x;
		int y = blockIdx.y * blockDim.y + threadIdx.y;
		if (x >= cols && y >= rows)
			return;

		float z = depth.ptr(y)[x];
		if (isnan(z) || z < DeviceMap::DepthMin ||
			z > DeviceMap::DepthMax)
			return;

		float thresh = DeviceMap::TruncateDist / 2;
		float z_near = min(DeviceMap::DepthMax, z - thresh);
		float z_far = min(DeviceMap::DepthMax, z + thresh);
		if (z_near >= z_far)
			return;

		float3 pt_near = unproject(x, y, z_near) * DeviceMap::voxelSizeInv;
		float3 pt_far = unproject(x, y, z_far) * DeviceMap::voxelSizeInv;
		float3 dir = pt_far - pt_near;

		float length = norm(dir);
		int nSteps = (int) ceil(2.0 * length);
		dir = dir / (float) (nSteps - 1);

		for (int i = 0; i < nSteps; ++i) {
			int3 blockPos = map.voxelPosToBlockPos(make_int3(pt_near));
			map.CreateBlock(blockPos);
			pt_near += dir;
		}
	}

	__device__ inline void CheckFullVisibility() {

		__shared__ bool bScan;
		if (threadIdx.x == 0)
			bScan = false;
		__syncthreads();
		uint val = 0;
		int x = blockDim.x * blockIdx.x + threadIdx.x;
		if (x < map.hashEntries.size) {
			HashEntry& e = map.hashEntries[x];
			if (e.ptr != EntryAvailable) {
				if (CheckBlockVisibility(e.pos)) {
					bScan = true;
					val = 1;
				}
			}
		}

		__syncthreads();
		if (bScan) {
			int offset = ComputeOffset<1024>(val, noVisibleBlocks);
			if (offset != -1 && offset < map.visibleEntries.size
					&& x < map.hashEntries.size)
				map.visibleEntries[offset] = map.hashEntries[x];
		}
	}

	__device__ inline void integrateColor() {

		if(blockIdx.x >= map.visibleEntries.size ||
		   blockIdx.x >= *noVisibleBlocks)
			return;

		HashEntry& entry = map.visibleEntries[blockIdx.x];
		if (entry.ptr == EntryAvailable)
			return;

		int3 block_pos = map.blockPosToVoxelPos(entry.pos);

		#pragma unroll
		for(int i = 0; i < 8; ++i) {
			int3 localPos = make_int3(threadIdx.x, threadIdx.y, i);
			int locId = map.localPosToLocalIdx(localPos);
			float3 pos = map.voxelPosToWorldPos(block_pos + localPos);
			pos = RviewInv * (pos - tview);
			int2 uv = make_int2(project(pos));
			if (uv.x < 0 || uv.y < 0 || uv.x >= cols || uv.y >= rows)
				continue;

			float dp = depth.ptr(uv.y)[uv.x];
			if (isnan(dp) || dp > maxDepth || dp < minDepth)
				continue;

			float thresh = DeviceMap::TruncateDist;
			float sdf = dp - pos.z;

			if (sdf >= -thresh) {

				sdf = fmin(1.0f, sdf / thresh);
				float4 nl = nmap.ptr(uv.y)[uv.x];
				if(isnan(nl.x))
					continue;

				float w = nl * normalised(make_float4(pos));
				w = 1;
				float3 val = make_float3(rgb.ptr(uv.y)[uv.x]);
				Voxel & prev = map.voxelBlocks[entry.ptr + locId];
				if(prev.weight == 0) {
					prev = Voxel(sdf, 1, make_uchar3(val));
				} else {
					val = val / 255.f;
					float3 old = make_float3(prev.color) / 255.f;
					float3 res = (w * 0.2f * val + (1 - w * 0.2f) * old) * 255.f;
					prev.sdf = (prev.sdf * prev.weight + w * sdf) / (prev.weight + w);
					prev.weight = min(255, prev.weight + 1);
					prev.color = make_uchar3(res);
				}
			}
		}
	}

	__device__ inline void deIntegrateColor() {

		if(blockIdx.x >= map.visibleEntries.size ||
		   blockIdx.x >= *noVisibleBlocks)
			return;

		HashEntry& entry = map.visibleEntries[blockIdx.x];
		if (entry.ptr == EntryAvailable)
			return;

		int3 block_pos = map.blockPosToVoxelPos(entry.pos);

		#pragma unroll
		for(int i = 0; i < 8; ++i) {
			int3 localPos = make_int3(threadIdx.x, threadIdx.y, i);
			int locId = map.localPosToLocalIdx(localPos);
			float3 pos = map.voxelPosToWorldPos(block_pos + localPos);
			pos = RviewInv * (pos - tview);
			int2 uv = make_int2(project(pos));
			if (uv.x < 0 || uv.y < 0 || uv.x >= cols || uv.y >= rows)
				continue;

			float dp = depth.ptr(uv.y)[uv.x];
			if (isnan(dp) || dp > maxDepth || dp < minDepth)
				continue;

			float thresh = DeviceMap::TruncateDist;
			float sdf = dp - pos.z;

			if (sdf >= -thresh) {

				sdf = fmin(1.0f, sdf / thresh);
				float4 nl = nmap.ptr(uv.y)[uv.x];
				if(isnan(nl.x))
					continue;

				float w = nl * normalised(make_float4(pos));
				w = 1;
				float3 val = make_float3(rgb.ptr(uv.y)[uv.x]);
				Voxel & prev = map.voxelBlocks[entry.ptr + locId];
				if(prev.weight == 0) {
					return;
				} else {
					val = val / 255.f;
					float3 old = make_float3(prev.color) / 255.f;
					float3 res = ((1 - w * 0.2f) * old - w * 0.2f * val) * 255.f;
					prev.sdf = (prev.sdf * prev.weight - w * sdf) / (prev.weight - w);
					prev.weight = max(0, prev.weight - 1);
					prev.color = make_uchar3(res);
				}
			}
		}
	}
};

__global__ void CreateBlocksKernel(Fusion fuse) {
	fuse.CreateBlocks();
}

__global__ void FuseColorKernal(Fusion fuse) {
	fuse.integrateColor();
}

__global__ void DefuseColorKernal(Fusion fuse) {
	fuse.deIntegrateColor();
}

__global__ void CheckVisibleBlockKernel(Fusion fuse) {
	fuse.CheckFullVisibility();
}

void CheckBlockVisibility(DeviceMap map,
					     DeviceArray<uint> & noVisibleBlocks,
						 Matrix3f Rview,
						 Matrix3f RviewInv,
						 float3 tview,
						 int cols,
						 int rows,
						 float fx,
						 float fy,
						 float cx,
						 float cy,
						 float depthMax,
						 float depthMin,
						 uint * host_data) {

	noVisibleBlocks.clear();

	Fusion fuse;
	fuse.map = map;
	fuse.Rview = Rview;
	fuse.RviewInv = RviewInv;
	fuse.tview = tview;
	fuse.fx = fx;
	fuse.fy = fy;
	fuse.cx = cx;
	fuse.cy = cy;
	fuse.invfx = 1.0 / fx;
	fuse.invfy = 1.0 / fy;
	fuse.rows = rows;
	fuse.cols = cols;
	fuse.noVisibleBlocks = noVisibleBlocks;
	fuse.maxDepth = depthMax;
	fuse.minDepth = depthMin;

	dim3 thread = dim3(1024);
	dim3 block = dim3(DivUp((int) DeviceMap::NumEntries, thread.x));

	CheckVisibleBlockKernel<<<block, thread>>>(fuse);

	host_data[0] = 0;
	noVisibleBlocks.download((void*) host_data);
	if (host_data[0] == 0)
		return;
}

void FuseMapColor(const DeviceArray2D<float> & depth,
				  const DeviceArray2D<uchar3> & color,
				  const DeviceArray2D<float4> & nmap,
				  DeviceArray<uint> & noVisibleBlocks,
				  Matrix3f Rview,
				  Matrix3f RviewInv,
				  float3 tview,
				  DeviceMap map,
				  float fx,
				  float fy,
				  float cx,
				  float cy,
				  float depthMax,
				  float depthMin,
				  uint * host_data) {

	int cols = depth.cols;
	int rows = depth.rows;
	noVisibleBlocks.clear();

	Fusion fuse;
	fuse.map = map;
	fuse.Rview = Rview;
	fuse.RviewInv = RviewInv;
	fuse.tview = tview;
	fuse.fx = fx;
	fuse.fy = fy;
	fuse.cx = cx;
	fuse.cy = cy;
	fuse.invfx = 1.0 / fx;
	fuse.invfy = 1.0 / fy;
	fuse.depth = depth;
	fuse.rgb = color;
	fuse.nmap = nmap;
	fuse.rows = rows;
	fuse.cols = cols;
	fuse.noVisibleBlocks = noVisibleBlocks;
	fuse.maxDepth = DeviceMap::DepthMax;
	fuse.minDepth = DeviceMap::DepthMin;

	dim3 thread(16, 8);
	dim3 block(DivUp(cols, thread.x), DivUp(rows, thread.y));

	CreateBlocksKernel<<<block, thread>>>(fuse);

	SafeCall(hipDeviceSynchronize());
	SafeCall(hipGetLastError());

	thread = dim3(1024);
	block = dim3(DivUp((int) DeviceMap::NumEntries, thread.x));

	CheckVisibleBlockKernel<<<block, thread>>>(fuse);

	SafeCall(hipDeviceSynchronize());
	SafeCall(hipGetLastError());

	host_data[0] = 0;
	noVisibleBlocks.download((void*) host_data);
	if (host_data[0] == 0)
		return;

	thread = dim3(8, 8);
	block = dim3(host_data[0]);

	FuseColorKernal<<<block, thread>>>(fuse);

	SafeCall(hipDeviceSynchronize());
	SafeCall(hipGetLastError());
}

void DefuseMapColor(const DeviceArray2D<float> & depth,
				  	const DeviceArray2D<uchar3> & color,
				  	const DeviceArray2D<float4> & nmap,
				  	DeviceArray<uint> & noVisibleBlocks,
				  	Matrix3f Rview,
				  	Matrix3f RviewInv,
				  	float3 tview,
				  	DeviceMap map,
				  	float fx,
				  	float fy,
				  	float cx,
				  	float cy,
				  	float depthMax,
				  	float depthMin,
				  	uint * host_data) {

	int cols = depth.cols;
	int rows = depth.rows;
	noVisibleBlocks.clear();

	Fusion fuse;
	fuse.map = map;
	fuse.Rview = Rview;
	fuse.RviewInv = RviewInv;
	fuse.tview = tview;
	fuse.fx = fx;
	fuse.fy = fy;
	fuse.cx = cx;
	fuse.cy = cy;
	fuse.invfx = 1.0 / fx;
	fuse.invfy = 1.0 / fy;
	fuse.depth = depth;
	fuse.rgb = color;
	fuse.nmap = nmap;
	fuse.rows = rows;
	fuse.cols = cols;
	fuse.noVisibleBlocks = noVisibleBlocks;
	fuse.maxDepth = DeviceMap::DepthMax;
	fuse.minDepth = DeviceMap::DepthMin;

	dim3 thread = dim3(1024);
	dim3 block = dim3(DivUp((int) DeviceMap::NumEntries, thread.x));

	CheckVisibleBlockKernel<<<block, thread>>>(fuse);

	SafeCall(hipDeviceSynchronize());
	SafeCall(hipGetLastError());

	host_data[0] = 0;
	noVisibleBlocks.download((void*) host_data);
	if (host_data[0] == 0)
		return;

	thread = dim3(8, 8);
	block = dim3(host_data[0]);

	DefuseColorKernal<<<block, thread>>>(fuse);

	SafeCall(hipDeviceSynchronize());
	SafeCall(hipGetLastError());
}

__global__ void ResetHashKernel(DeviceMap map) {

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if(x < map.hashEntries.size) {
		map.hashEntries[x].release();
		map.visibleEntries[x].release();
	}

	if (x < DeviceMap::NumBuckets) {
		map.bucketMutex[x] = EntryAvailable;
	}
}

__global__ void ResetSdfBlockKernel(DeviceMap map) {

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if(x < DeviceMap::NumSdfBlocks) {
		map.heapMem[x] = DeviceMap::NumSdfBlocks - x - 1;
	}

	int blockIdx = x * DeviceMap::BlockSize3;
	for(int i = 0; i < DeviceMap::BlockSize3; ++i, ++blockIdx) {
		map.voxelBlocks[blockIdx].release();
	}

	if(x == 0) {
		map.heapCounter[0] = DeviceMap::NumSdfBlocks - 1;
		map.entryPtr[0] = 1;
	}
}

void ResetMap(DeviceMap map) {

	dim3 thread(1024);
	dim3 block(DivUp((int) DeviceMap::NumEntries, thread.x));

	ResetHashKernel<<<block, thread>>>(map);

	block = dim3(DivUp((int) DeviceMap::NumSdfBlocks, thread.x));
	ResetSdfBlockKernel<<<block, thread>>>(map);

	SafeCall(hipDeviceSynchronize());
	SafeCall(hipGetLastError());
}

__global__ void ResetKeyPointsKernel(KeyMap map) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	map.ResetKeys(x);
}

void ResetKeyPoints(KeyMap map) {

	dim3 thread(1024);
	dim3 block(DivUp((int) KeyMap::maxEntries, thread.x));

	ResetKeyPointsKernel<<<block, thread>>>(map);

	SafeCall(hipDeviceSynchronize());
	SafeCall(hipGetLastError());
}

struct KeyFusion {

	__device__ __forceinline__ void CollectKeys() {

		__shared__ bool scan;
		if(threadIdx.x == 0)
			scan = false;
		__syncthreads();

		uint val = 0;
		int x = blockDim.x * blockIdx.x + threadIdx.x;
		if(x < map.Keys.size) {
			SURF * key = &map.Keys[x];
			if(key->valid) {
				scan = true;
				val = 1;
			}
		}
		__syncthreads();

		if(scan) {
			int offset = ComputeOffset<1024>(val, nokeys);
			if(offset > 0 && x < map.Keys.size) {
				memcpy(&keys[offset], &map.Keys[x], sizeof(SURF));
			}
		}
	}

	__device__ __forceinline__ void InsertKeys() {

		int x = blockDim.x * blockIdx.x + threadIdx.x;
		if (x < size)
			map.InsertKey(&keys[x], index[x]);
	}

	KeyMap map;

	uint * nokeys;

	PtrSz<SURF> keys;

	size_t size;

	PtrSz<int> index;
};

__global__ void CollectKeyPointsKernel(KeyFusion fuse) {
	fuse.CollectKeys();
}

__global__ void InsertKeyPointsKernel(KeyFusion fuse) {
	fuse.InsertKeys();
}

void CollectKeyPoints(KeyMap map, DeviceArray<SURF> & keys, DeviceArray<uint> & noKeys) {

	KeyFusion fuse;
	fuse.map = map;
	fuse.keys = keys;
	fuse.nokeys = noKeys;

	dim3 thread(1024);
	dim3 block(DivUp(map.Keys.size, thread.x));

	CollectKeyPointsKernel<<<block, thread>>>(fuse);

	SafeCall(hipDeviceSynchronize());
	SafeCall(hipGetLastError());
}

void InsertKeyPoints(KeyMap map, DeviceArray<SURF> & keys,
		DeviceArray<int> & keyIndex, size_t size) {

	if(size == 0)
		return;

	KeyFusion fuse;

	fuse.map = map;
	fuse.keys = keys;
	fuse.size = size;
	fuse.index = keyIndex;

	dim3 thread(1024);
	dim3 block(DivUp(size, thread.x));

	InsertKeyPointsKernel<<<block, thread>>>(fuse);

	SafeCall(hipDeviceSynchronize());
	SafeCall(hipGetLastError());
}
